#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "crystal.cuh"
#include "test_util.h"
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>

constexpr int32_t block_threads    = 128;
constexpr int32_t items_per_thread = 4;
constexpr int32_t seed             = 0;
// Possible selectivities: 50%, 33%, 25%, 20%, 10%, 5%
thrust::host_vector<int32_t> possible_inverse_selectivities = {2, 3, 4, 5, 10, 20, 40};

//--------------------------------------------------------------------------------------------------
// Functors
//--------------------------------------------------------------------------------------------------
struct SelectOp : thrust::unary_function<int32_t, bool>
{
  int32_t mod;

  __host__ __device__ explicit SelectOp(int32_t mod)
      : mod{mod}
  {}

  __host__ __device__ __forceinline__ bool operator()(const int32_t& value) const
  {
    return (value % mod) == 0;
  }
};

//--------------------------------------------------------------------------------------------------
// Kernels
//--------------------------------------------------------------------------------------------------
template <typename InputIt, typename StencilIt, typename Predicate, typename OutputIt>
__global__ void UnorderedSelectionKernel(InputIt input,
                                         StencilIt stencil,
                                         Predicate predicate,
                                         int32_t num_in,
                                         OutputIt output,
                                         int32_t* num_out)
{
  // Typedefs
  typedef typename thrust::iterator_traits<InputIt>::value_type InputT;
  typedef typename thrust::iterator_traits<StencilIt>::value_type StencilT;
  typedef hipcub::BlockLoad<InputT, block_threads, items_per_thread, hipcub::BLOCK_LOAD_STRIPED>
    BlockLoadInput;
  typedef hipcub::BlockLoad<StencilT, block_threads, items_per_thread, hipcub::BLOCK_LOAD_STRIPED>
    BlockLoadStencil;
  typedef hipcub::BlockScan<int32_t, block_threads> BlockScan;
  typedef crystal::
    BlockFlag<int32_t, block_threads, items_per_thread, crystal::DataArrangement::Striped>
      BlockFlag;
  typedef crystal::BlockShuffle<InputT, block_threads, items_per_thread> BlockShuffle;
  typedef crystal::KernelConfig<block_threads, items_per_thread> KernelConfig;

  // Shared memory
  __shared__ typename BlockLoadInput::TempStorage temp_load_input_storage;
  __shared__ typename BlockLoadStencil::TempStorage temp_load_stencil_storage;
  __shared__ typename BlockScan::TempStorage temp_scan_storage;
  __shared__ typename BlockShuffle::TempStorage temp_shuffle_storage;
  __shared__ int32_t write_offset;

  // Thread memory
  InputT thread_input[items_per_thread];
  StencilT thread_stencil[items_per_thread];
  int32_t thread_flags[items_per_thread];
  int32_t prefix_sums[items_per_thread];
  int32_t num_selected = 0;
  KernelConfig kernel_config(blockIdx.x, num_in);

  // Do stuff
  if (!kernel_config.is_last_tile)
  {
    BlockLoadInput(temp_load_input_storage).Load(input + kernel_config.block_offset, thread_input);
    BlockLoadStencil(temp_load_stencil_storage)
      .Load(stencil + kernel_config.block_offset, thread_stencil);
    BlockFlag::SetFlags(thread_stencil, predicate, thread_flags);
  }
  else
  {
    BlockLoadInput(temp_load_input_storage)
      .Load(input + kernel_config.block_offset, thread_input, kernel_config.num_tile_items);
    BlockLoadStencil(temp_load_stencil_storage)
      .Load(stencil + kernel_config.block_offset, thread_stencil, kernel_config.num_tile_items);
    BlockFlag::InitFlags(thread_flags);
    BlockFlag::SetFlags(thread_stencil, predicate, thread_flags, kernel_config.num_tile_items);
  }
  BlockScan(temp_scan_storage).ExclusiveSum(thread_flags, prefix_sums, num_selected);
  if (threadIdx.x == 0)
  {
    write_offset = atomicAdd(num_out, num_selected);
  }
  __syncthreads(); // For write_offset
  BlockShuffle(temp_shuffle_storage)
    .ShuffleStore<crystal::DataArrangement::Striped>(output + write_offset,
                                                     thread_input,
                                                     thread_flags,
                                                     prefix_sums,
                                                     num_selected);
}

template <typename InputIt, typename OffsetIt, typename OutputIt>
__global__ void GatherKernel(InputIt input, OffsetIt offsets, OutputIt output, int32_t num_in)
{
  typedef typename thrust::iterator_traits<InputIt>::value_type InputT;
  typedef typename thrust::iterator_traits<OutputIt>::value_type OffsetT;
  typedef crystal::
    BlockLoad<OffsetT, block_threads, items_per_thread, crystal::DataArrangement::Striped>
      BlockLoadOffsets;
  typedef crystal::
    BlockLoad<InputT, block_threads, items_per_thread, crystal::DataArrangement::Striped>
      BlockLoadInputs;
  typedef crystal::
    BlockStore<InputT, block_threads, items_per_thread, crystal::DataArrangement::Striped>
      BlockStore;
  typedef crystal::KernelConfig<block_threads, items_per_thread> KernelConfig;

  // Thread memory
  InputT thread_input[items_per_thread];
  OffsetT thread_offsets[items_per_thread];
  KernelConfig kernel_config(blockIdx.x, num_in);

  if (!kernel_config.is_last_tile)
  {
    BlockLoadOffsets::Load(offsets + kernel_config.block_offset, thread_offsets);
    BlockLoadInputs::Gather(input, thread_offsets, thread_input);
    BlockStore::Store(output + kernel_config.block_offset, thread_input);
  }
  else
  {
    BlockLoadOffsets::Load(offsets + kernel_config.block_offset,
                           thread_offsets,
                           kernel_config.num_tile_items);
    BlockLoadInputs::Gather(input, thread_offsets, thread_input, kernel_config.num_tile_items);
    BlockStore::Store(output + kernel_config.block_offset,
                      thread_input,
                      kernel_config.num_tile_items);
  }
}

//--------------------------------------------------------------------------------------------------
// Sweep functions
//--------------------------------------------------------------------------------------------------
void SweepSelectivity(int32_t max_inverse_selectivity, int32_t output_size)
{
  // Initialize output buffers
  thrust::device_vector<int32_t> row_ids_out_sorted(output_size);
  thrust::device_vector<int32_t> row_ids_out_unsorted(output_size);
  thrust::device_vector<int32_t> row_ids_out_random(output_size);
  thrust::host_vector<int32_t> row_ids_out_random_host(output_size);
  thrust::device_vector<int32_t> gathered_data_sorted(output_size);
  thrust::device_vector<int32_t> gathered_data_unsorted(output_size);
  thrust::device_vector<int32_t> gathered_data_random(output_size);

  // Initialize row ids
  thrust::device_vector<int32_t> row_ids_in(output_size * max_inverse_selectivity);
  thrust::sequence(row_ids_in.begin(), row_ids_in.end(), 0);

  // Initialize/seed random generator
  std::default_random_engine rng(seed);

  // Loop over selectivities
  auto max_possible_inverse_selectivity =
    std::max_element(possible_inverse_selectivities.begin(), possible_inverse_selectivities.end());
  auto inverse_selectivity_iter = possible_inverse_selectivities.begin();
  while (inverse_selectivity_iter < max_possible_inverse_selectivity &&
         *inverse_selectivity_iter <= max_inverse_selectivity)
  {
    int32_t inverse_selectivity = *inverse_selectivity_iter;
    std::cout << "Selectivity: " << (1 / static_cast<float>(inverse_selectivity)) << "\n";

    // Initialize and populate stencil
    int32_t num_input = inverse_selectivity * output_size;
    thrust::host_vector<int32_t> stencil_host(num_input);
    thrust::device_vector<int32_t> stencil(num_input);
    thrust::sequence(stencil_host.begin(), stencil_host.end(), 0);
    std::shuffle(stencil_host.begin(), stencil_host.end(), rng);
    thrust::copy(stencil_host.begin(), stencil_host.end(), stencil.begin());
    CubDebugExit(hipDeviceSynchronize());

    // Initialize counters
    thrust::device_vector<int32_t> num_out_unsorted(1);
    thrust::device_vector<int32_t> num_out_sorted(1);

    // Cub order-preserving compaction
    uint8_t* temp_storage     = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::FlaggedIf(temp_storage,
                                 temp_storage_bytes,
                                 row_ids_in.begin(),
                                 stencil.begin(),
                                 row_ids_out_sorted.begin(),
                                 num_out_sorted.begin(),
                                 num_input,
                                 SelectOp{inverse_selectivity});
    CubDebugExit(hipMalloc(&temp_storage, temp_storage_bytes));
    hipcub::DeviceSelect::FlaggedIf(temp_storage,
                                 temp_storage_bytes,
                                 row_ids_in.begin(),
                                 stencil.begin(),
                                 row_ids_out_sorted.begin(),
                                 num_out_sorted.begin(),
                                 num_input,
                                 SelectOp{inverse_selectivity});
    CubDebugExit(hipDeviceSynchronize());

    // Crystal non-order-preserving compaction
    int32_t blocks_in_grid = hipcub::DivideAndRoundUp(num_input, block_threads * items_per_thread);
    UnorderedSelectionKernel<<<blocks_in_grid, block_threads>>>(
      row_ids_in.begin(),
      stencil.begin(),
      SelectOp{inverse_selectivity},
      num_input,
      row_ids_out_unsorted.begin(),
      thrust::raw_pointer_cast(num_out_unsorted.data()));
    CubDebugExit(hipDeviceSynchronize());

    // Generate completely random sequence of row ids
    thrust::copy(row_ids_out_sorted.begin(),
                 row_ids_out_sorted.end(),
                 row_ids_out_random_host.begin());
    std::shuffle(row_ids_out_random_host.begin(), row_ids_out_random_host.end(), rng);
    thrust::copy(row_ids_out_random_host.begin(),
                 row_ids_out_random_host.end(),
                 row_ids_out_random.begin());

    // Sanity check that the outputs are the same size
    if (num_out_sorted[0] != num_out_unsorted[0] || num_out_sorted[0] != output_size)
    {
      std::cerr << "Kernels produced inconsistent output sizes (cub / crystal / expected): "
                << num_out_sorted[0] << " / " << num_out_unsorted[0] << " / " << output_size
                << "\n";
      exit(EXIT_FAILURE);
    }

    // Thrust gathers
    auto sorted_gather_iter =
      thrust::make_permutation_iterator(stencil.begin(), row_ids_out_sorted.begin());
    auto unsorted_gather_iter =
      thrust::make_permutation_iterator(stencil.begin(), row_ids_out_unsorted.begin());
    thrust::copy(sorted_gather_iter,
                 sorted_gather_iter + num_out_sorted[0],
                 gathered_data_sorted.begin());
    CubDebugExit(hipDeviceSynchronize());
    thrust::copy(unsorted_gather_iter,
                 unsorted_gather_iter + num_out_unsorted[0],
                 gathered_data_unsorted.begin());
    CubDebugExit(hipDeviceSynchronize());

    // Custom gathers
    int32_t blocks_in_gather_grid =
      hipcub::DivideAndRoundUp(output_size, block_threads * items_per_thread);
    GatherKernel<<<blocks_in_gather_grid, block_threads>>>(stencil.begin(),
                                                           row_ids_out_sorted.begin(),
                                                           gathered_data_sorted.begin(),
                                                           output_size);
    CubDebugExit(hipDeviceSynchronize());
    GatherKernel<<<blocks_in_gather_grid, block_threads>>>(stencil.begin(),
                                                           row_ids_out_unsorted.begin(),
                                                           gathered_data_unsorted.begin(),
                                                           output_size);
    CubDebugExit(hipDeviceSynchronize());
    GatherKernel<<<blocks_in_gather_grid, block_threads>>>(stencil.begin(),
                                                           row_ids_out_random.begin(),
                                                           gathered_data_random.begin(),
                                                           output_size);
    CubDebugExit(hipDeviceSynchronize());

    // Free explicitly allocated resources
    if (temp_storage)
    {
      CubDebugExit(hipFree(temp_storage));
    }

    // Increment iterator
    ++inverse_selectivity_iter;
  }
}

//--------------------------------------------------------------------------------------------------
// Main
//--------------------------------------------------------------------------------------------------
int main(int argc, char** argv)
{
  // Gather command-line args
  int32_t max_inverse_selectivity = 0;
  int32_t output_size             = 1 << 22;
  CommandLineArgs args(argc, argv);
  args.GetCmdLineArgument("mis", max_inverse_selectivity);
  args.GetCmdLineArgument("os", output_size);

  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    std::cout << argv[0]
              << "\n\t[--mis=<max inverse selectivity, i.e. sweep selectivities 1/2...1/ms>]"
              << "\n\t[--os=<output size for compaction>]\n";
    return 0;
  }

  // Sweep selectivities
  if (max_inverse_selectivity > 0)
  {
    SweepSelectivity(max_inverse_selectivity, output_size);
  }

  return 0;
}